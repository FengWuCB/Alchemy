#include "hip/hip_runtime.h"
#include <>
#include "relu_layer.hpp"

namespace z {

template <typename T>
__global__ void relu_kernel(const size_t size, const T* InputData, double alpha, T* OutputData)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
        OutputData[i] = InputData[i] > (T)0.0 ? InputData[i] : alpha * InputData[i];
    }
}

template<typename T>
void ReLuLayer<T>::ForwardGPU(const vector<container_type *> &input,
                              const vector<container_type *> &output)
{
    auto count = input[0]->count();
    auto input_data = input[0]->gpu_data();
    auto output_data = output[0]->gpu_data();

    relu_kernel<<<CUDA_BLOCK_NUM(count), CUDA_THREAD_NUM>>>(count, input_data, relu_param_.alpha(), output_data);
}


template <typename T>
__global__ void drelu_kernel(const size_t size, const T* InputData, const T* OutputDiff, double alpha, T* InputDiff)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
        InputDiff[i] = OutputDiff[i] * ((InputData[i] > 0) ? 1 : alpha);
    }
}

template<typename T>
void ReLuLayer<T>::BackwardGPU(const vector<container_type *> &input,
                               const vector<container_type *> &output)
{
    auto count = input[0]->count();
    auto input_data = input[0]->gpu_data();
    auto input_diff = input[0]->gpu_diff();
    auto output_diff = output[0]->gpu_diff();

    drelu_kernel<<<CUDA_BLOCK_NUM(count), CUDA_THREAD_NUM>>>(count, input_data, output_diff, relu_param_.alpha(), input_diff);
}

template void ReLuLayer<float>::ForwardGPU(const vector<container_type *> &input, const vector<container_type *> &output);
template void ReLuLayer<double>::ForwardGPU(const vector<container_type *> &input, const vector<container_type *> &output);
template void ReLuLayer<float>::BackwardGPU(const vector<container_type *> &input, const vector<container_type *> &output);
template void ReLuLayer<double>::BackwardGPU(const vector<container_type *> &input, const vector<container_type *> &output);
}