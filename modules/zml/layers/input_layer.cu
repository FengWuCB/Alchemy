#include "input_layer.hpp"

namespace z {

template<typename T>
void InputLayer<T>::ForwardGPU(const vector<container_type *> &input,
                                const vector<container_type *> &output)
{
    /// data
    auto data_ptr = output[0]->gpu_data();
    auto data_count = data_[0].first.size();
    auto data_size = data_count * sizeof(T);

    /// label
    auto label_ptr = output[1]->gpu_data();
    auto label_size = 10 * sizeof(T);

    for(size_t i = 0; i < input_param_.batch_size(); ++i, ++index_) {
        index_ %= data_num_;
        if(!index_) shuffle();

        auto& item = data_[index_];

        const auto& image = _Matrix<T>(item.first) * input_param_.scale();
        hipMemcpy(data_ptr, image.data, data_size, hipMemcpyHostToDevice);
//        memmove(data_ptr, image.data, data_size);
        data_ptr += data_count;

        _Matrix<T> temp(10, 1, 1, (T)0);
        temp.at(item.second) = 1;
        hipMemcpy(label_ptr, temp.data, label_size, hipMemcpyHostToDevice);
//        memmove(label_ptr, temp.data, label_size);
        label_ptr += 10;
    }
}

template void InputLayer<float>::ForwardGPU(const vector<container_type *> &input, const vector<container_type *> &output);
template void InputLayer<double>::ForwardGPU(const vector<container_type *> &input, const vector<container_type *> &output);
}