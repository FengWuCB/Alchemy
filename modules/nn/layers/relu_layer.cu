#include "hip/hip_runtime.h"
#include "relu_layer.h"
#include <>

namespace alchemy {

template <typename T>
__global__ void relu_kernel(const size_t size, const T* InputData, double alpha, T* OutputData)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
        OutputData[i] = InputData[i] > (T)0.0 ? InputData[i] : alpha * InputData[i];
    }
}

template<typename T>
void ReLuLayer<T>::ForwardGPU(const vector<Blob<T> *> &input,
                              const vector<Blob<T> *> &output)
{
    auto count = input[0]->count();
    auto input_data = input[0]->data_gptr();
    auto output_data = output[0]->data_gptr();

    relu_kernel<<<CUDA_BLOCK_NUM(count), CUDA_THREAD_NUM>>>(count, input_data, relu_param_.alpha(), output_data);
}


template <typename T>
__global__ void drelu_kernel(const size_t size, const T* InputData, const T* OutputDiff, double alpha, T* InputDiff)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
        InputDiff[i] = OutputDiff[i] * ((InputData[i] > 0) ? 1 : alpha);
    }
}

template<typename T>
void ReLuLayer<T>::BackwardGPU(const vector<Blob<T> *> &input,
                               const vector<Blob<T> *> &output)
{
    auto count = input[0]->count();
    auto input_data = input[0]->data_gptr();
    auto input_diff = input[0]->diff_gptr();
    auto output_diff = output[0]->diff_gptr();

    drelu_kernel<<<CUDA_BLOCK_NUM(count), CUDA_THREAD_NUM>>>(count, input_data, output_diff, relu_param_.alpha(), input_diff);
}

template void ReLuLayer<float>::ForwardGPU(const vector<Blob<float> *> &input, const vector<Blob<float> *> &output);
template void ReLuLayer<double>::ForwardGPU(const vector<Blob<double> *> &input, const vector<Blob<double> *> &output);
template void ReLuLayer<float>::BackwardGPU(const vector<Blob<float> *> &input, const vector<Blob<float> *> &output);
template void ReLuLayer<double>::BackwardGPU(const vector<Blob<double> *> &input, const vector<Blob<double> *> &output);
}