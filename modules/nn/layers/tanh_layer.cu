#include "hip/hip_runtime.h"
#include "tanh_layer.h"
#include <>

namespace alchemy {

template <typename T>
__global__ void tanh_kernel(const size_t size, const T* A, T* B)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
        B[i] = std::tanh(A[i]);
    }
}
template<typename T>
void TanhLayer<T>::ForwardGPU(const vector<Blob<T> *> &input,
                              const vector<Blob<T> *> &output)
{
    const auto count = input[0]->count();
    const auto input_data = input[0]->data_gptr();
    auto output_data = output[0]->mutable_data_gptr();

    tanh_kernel<<<CUDA_BLOCK_NUM(count), CUDA_THREAD_NUM>>>(count, input_data, output_data);
}

template <typename T>
__global__ void dtanh_kernel(const size_t size, const T* OutputData, const T* OutputDiff, T* InputDiff)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
        auto tx = OutputData[i];
        InputDiff[i] = OutputDiff[i] *(1 - tx * tx);
    }
}
template<typename T>
void TanhLayer<T>::BackwardGPU(const vector<Blob<T> *> &input,
                               const vector<Blob<T> *> &output)
{
    auto count = input[0]->count();
    auto output_data = output[0]->data_gptr();
    auto output_diff = output[0]->diff_gptr();
    auto input_diff = input[0]->mutable_diff_gptr();

    dtanh_kernel<<<CUDA_BLOCK_NUM(count), CUDA_THREAD_NUM>>>(count, output_data, output_diff, input_diff);
}

template void TanhLayer<float>::ForwardGPU(const vector<Blob<float> *> &input, const vector<Blob<float> *> &output);
template void TanhLayer<double>::ForwardGPU(const vector<Blob<double> *> &input, const vector<Blob<double> *> &output);
template void TanhLayer<float>::BackwardGPU(const vector<Blob<float> *> &input, const vector<Blob<float> *> &output);
template void TanhLayer<double>::BackwardGPU(const vector<Blob<double> *> &input, const vector<Blob<double> *> &output);
}