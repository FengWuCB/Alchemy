#include <math/math_op.h>
#include "input_layer.h"

namespace alchemy {

template<typename T>
void InputLayer<T>::ForwardGPU(const vector<Blob<T> *> &input,
                                const vector<Blob<T> *> &output)
{
    auto batch_size = input_param_.batch_size();
    /// data
    auto images_ptr = data_.images().get();
    hipMemcpy(output[0]->data_gptr(),
               images_ptr + index_ * data_.image_size(),
               batch_size * data_.image_size() * sizeof(T),
               hipMemcpyHostToDevice);

    /// label
    auto labels_ptr = data_.labels().get();
    hipMemcpy(output[1]->data_gptr(),
               labels_ptr + index_ * data_.label_size(),
               batch_size * data_.label_size() * sizeof(T),
               hipMemcpyHostToDevice);

    index_ = (index_ + batch_size) % data_num_;
    if(data_num_ - index_ < batch_size) index_ = 0;
}

template void InputLayer<float>::ForwardGPU(const vector<Blob<float> *> &input, const vector<Blob<float> *> &output);
template void InputLayer<double>::ForwardGPU(const vector<Blob<double> *> &input, const vector<Blob<double> *> &output);
}